#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"


#define INF 0x3f3f3f3f
#define THREADS_PER_BLOCK_SIDE 16

using namespace std;

__global__ void calculate(int *input, int *output, int n, int k)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n && j < n){
        if (input[i * n + k] + input[k * n + j] < input[i * n + j])
            input[i * n + j] = input[i * n + k] + input[k * n + j];
        output[i * n + j] = input[i * n + j];
    }
}

void writeToFile(int *output, int n)
{
    ofstream fout("royfloyd.out");
    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            if(output[i * n + j] == INF)
                output[i * n + j] = 0;
            fout << output[i * n + j] << " ";
        }
        fout << "\n";
    }
}

int main()
{
    int n;
    int *graphHostInput;
    int *graphHostOutput;
    int *graphDeviceInput;
    int *graphDeviceOutput;

    ifstream fin("royfloyd.in");
    fin >> n;

    graphHostInput = (int*) malloc(sizeof(int) * n * n);

    for(int i = 0;i < n;i++)
        for(int j = 0;j < n;j++)
        {
            fin >> graphHostInput[i * n + j];
            if(i != j && !graphHostInput[i * n + j])
                graphHostInput[i * n + j] = INF;
        }

    graphHostOutput = (int*) malloc(sizeof(int) * n * n);
    hipMalloc((void**)&graphDeviceInput, sizeof(int) * n * n);
    hipMalloc((void**)&graphDeviceOutput, sizeof(int) * n * n);

    hipMemcpy(graphDeviceInput, graphHostInput, n * n * sizeof(int), hipMemcpyHostToDevice);
    const int BLOCKS_PER_GRAPH_SIDE = ((n + THREADS_PER_BLOCK_SIDE - 1) / THREADS_PER_BLOCK_SIDE);
    dim3 blocks(BLOCKS_PER_GRAPH_SIDE, BLOCKS_PER_GRAPH_SIDE, 1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE, 1);
    for (int k = 0; k < n; k++) {
        calculate<<<blocks, threadsPerBlock>>>(graphDeviceInput, graphDeviceOutput, n, k);
    }
    hipMemcpy(graphHostOutput, graphDeviceOutput, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    hipFree(graphDeviceInput);
    hipFree(graphDeviceOutput);

    writeToFile(graphHostOutput, n);

    free(graphHostInput);
    free(graphHostOutput);
    return 0;
}